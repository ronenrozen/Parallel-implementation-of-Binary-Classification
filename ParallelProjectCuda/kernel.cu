#include "hip/hip_runtime.h"
#include "Header.h"
#include <stdlib.h>
#define MAX_NUM_OF_THREADS 1000


__device__ int atomicAdd(int* address, int val);

int * nMiss;

/**********************************************
Algorithm function cuda version
***********************************************/
__device__ double fOnGPU(Model * mod, double * points, int K) {
	double sum = mod->bias;
	for (int i = 0; i < K; i++)
	{
		sum += mod->weights[i] * points[i];
	}
	return SIGN(sum);
}

/**********************************************
Each thread calculates different element of points
***********************************************/
__global__ void calculatePointsOnKernel(Vector *points, int numOfthreads, Model * mod, int K, int* nMiss) {
	int thread_index = threadIdx.x;
	int block_index = blockIdx.x;
	int index = thread_index + block_index * numOfthreads;

	int prediction = fOnGPU(mod, points[index].points, K);
	if (points[index].expected != prediction)
	{
		atomicAdd(nMiss, 1);
	}
}



/**********************************************
init the vectors for dev_results and dev_points
doing it once per process and not in every train
***********************************************/
hipError_t initCuda(int numOfTasks, Vector **dev_points, Model **dev_mod, Vector *points) {
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
	}

	// Allocate GPU buffer for points.
	cudaStatus = hipMalloc((void**)dev_points, MAX_POINTS * sizeof(Vector));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc for tasks failed!");
		return cudaStatus;
	}

	// Copy points from CPU to GPU.
	cudaStatus = hipMemcpy(*dev_points, points, MAX_POINTS * sizeof(Vector), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy for tasks failed!");
		return cudaStatus;
	}

	// Allocate GPU buffer for Model.
	cudaStatus = hipMalloc((void**)dev_mod, sizeof(Model));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc for dev_mod failed!");
		return hipErrorUnknown;
	}
	// Allocate GPU buffer for nMiss.
	cudaStatus = hipMalloc((void**)&nMiss, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc for nMiss failed!");
		return hipErrorUnknown;
	}
	return cudaStatus;
}

/**********************************************
Main func of cuda, here we manage all the operations of the cuda

***********************************************/
hipError_t calculateWithCuda(Vector *dev_points, int numOfPoints, double *q, Model * mod, Model * dev_mod, int K)
{

	hipError_t cudaStatus;

	// Copy model from CPU to GPU.
	cudaStatus = hipMemcpy(dev_mod, mod, sizeof(Model), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy for tasks failed!");
		return cudaStatus;
	}

	int blocks = numOfPoints / MAX_NUM_OF_THREADS > 1 ? numOfPoints / MAX_NUM_OF_THREADS : 1;
	int threads = numOfPoints / MAX_NUM_OF_THREADS > 1 ? MAX_NUM_OF_THREADS : numOfPoints;

	// Launch a kernel on the GPU with one thread for each element.
	calculatePointsOnKernel << <blocks, threads >> > (dev_points, threads, dev_mod, K, nMiss);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "resultKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching resultKernel!\n", cudaStatus);
		return cudaStatus;
	}

	int * nMissCPU = (int *)malloc(sizeof(int));
	// Copy nMiss from GPU buffer to CPU memory.
	cudaStatus = hipMemcpy(nMissCPU, nMiss, sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy for final result failed!");
		return cudaStatus;
	}
	*q = (double)*nMissCPU / (double)numOfPoints;
	return cudaStatus;
}
